#include "hip/hip_runtime.h"
﻿#include "kernel_common.cuh"

namespace VLR {
    template <typename T>
    RT_FUNCTION T* getData(uint32_t nodeDescIndex) {
        return pv_smallNodeDescriptorBuffer[nodeDescIndex].getData<T>();
    }



    RT_CALLABLE_PROGRAM Point3D GeometryShaderNode_Point3D(const ShaderNodeSocket &socket,
                                                           const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        return surfPt.position;
    }

    RT_CALLABLE_PROGRAM Normal3D GeometryShaderNode_Normal3D(const ShaderNodeSocket &socket,
                                                             const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        if (socket.option == 0)
            return surfPt.geometricNormal;
        else if (socket.option == 1)
            return surfPt.shadingFrame.z;
        return Normal3D(0, 0, 0);
    }

    RT_CALLABLE_PROGRAM Vector3D GeometryShaderNode_Vector3D(const ShaderNodeSocket &socket,
                                                             const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        if (socket.option == 0)
            return surfPt.shadingFrame.x;
        else if (socket.option == 1)
            return surfPt.shadingFrame.y;
        return Vector3D::Zero();
    }

    RT_CALLABLE_PROGRAM Point3D GeometryShaderNode_TextureCoordinates(const ShaderNodeSocket &socket,
                                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        return Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0);
    }



    RT_CALLABLE_PROGRAM float Float2ShaderNode_float(const ShaderNodeSocket &socket,
                                                     const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float2ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return calcNode(nodeData.node0, nodeData.imm0, surfPt, wls);
        else if (socket.option == 1)
            return calcNode(nodeData.node1, nodeData.imm1, surfPt, wls);
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Float2ShaderNode_float2(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float2ShaderNode>(socket.nodeDescIndex);
        return optix::make_float2(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                  calcNode(nodeData.node1, nodeData.imm1, surfPt, wls));
    }



    template <>
    RT_FUNCTION Float3ShaderNode* getData<Float3ShaderNode>(uint32_t nodeDescIndex) {
        return pv_mediumNodeDescriptorBuffer[nodeDescIndex].getData<Float3ShaderNode>();
    }
    
    RT_CALLABLE_PROGRAM float Float3ShaderNode_float(const ShaderNodeSocket &socket,
                                                     const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float3ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return calcNode(nodeData.node0, nodeData.imm0, surfPt, wls);
        else if (socket.option == 1)
            return calcNode(nodeData.node1, nodeData.imm1, surfPt, wls);
        else if (socket.option == 2)
            return calcNode(nodeData.node2, nodeData.imm2, surfPt, wls);
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Float3ShaderNode_float2(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float3ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return optix::make_float2(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                      calcNode(nodeData.node1, nodeData.imm1, surfPt, wls));
        else if (socket.option == 1)
            return optix::make_float2(calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                      calcNode(nodeData.node2, nodeData.imm2, surfPt, wls));
        return optix::make_float2(0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float3 Float3ShaderNode_float3(const ShaderNodeSocket &socket, 
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float3ShaderNode>(socket.nodeDescIndex);
        return optix::make_float3(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                  calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                  calcNode(nodeData.node2, nodeData.imm2, surfPt, wls));
    }



    template <>
    RT_FUNCTION Float4ShaderNode* getData<Float4ShaderNode>(uint32_t nodeDescIndex) {
        return pv_mediumNodeDescriptorBuffer[nodeDescIndex].getData<Float4ShaderNode>();
    }
    
    RT_CALLABLE_PROGRAM float Float4ShaderNode_float(const ShaderNodeSocket &socket,
                                                     const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float4ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return calcNode(nodeData.node0, nodeData.imm0, surfPt, wls);
        else if (socket.option == 1)
            return calcNode(nodeData.node1, nodeData.imm1, surfPt, wls);
        else if (socket.option == 2)
            return calcNode(nodeData.node2, nodeData.imm2, surfPt, wls);
        else if (socket.option == 3)
            return calcNode(nodeData.node3, nodeData.imm3, surfPt, wls);
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Float4ShaderNode_float2(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float4ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return optix::make_float2(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                      calcNode(nodeData.node1, nodeData.imm1, surfPt, wls));
        else if (socket.option == 1)
            return optix::make_float2(calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                      calcNode(nodeData.node2, nodeData.imm2, surfPt, wls));
        else if (socket.option == 2)
            return optix::make_float2(calcNode(nodeData.node2, nodeData.imm2, surfPt, wls),
                                      calcNode(nodeData.node3, nodeData.imm3, surfPt, wls));
        return optix::make_float2(0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float3 Float4ShaderNode_float3(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float4ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return optix::make_float3(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                      calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                      calcNode(nodeData.node2, nodeData.imm2, surfPt, wls));
        else if (socket.option == 1)
            return optix::make_float3(calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                      calcNode(nodeData.node2, nodeData.imm2, surfPt, wls),
                                      calcNode(nodeData.node3, nodeData.imm3, surfPt, wls));
        return optix::make_float3(0.0f, 0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float4 Float4ShaderNode_float4(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float4ShaderNode>(socket.nodeDescIndex);
        return optix::make_float4(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                  calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                  calcNode(nodeData.node2, nodeData.imm2, surfPt, wls),
                                  calcNode(nodeData.node3, nodeData.imm3, surfPt, wls));
    }



    template <>
    RT_FUNCTION ScaleAndOffsetFloatShaderNode* getData<ScaleAndOffsetFloatShaderNode>(uint32_t nodeDescIndex) {
        return pv_mediumNodeDescriptorBuffer[nodeDescIndex].getData<ScaleAndOffsetFloatShaderNode>();
    }
    
    RT_CALLABLE_PROGRAM float ScaleAndOffsetFloatShaderNode_float(const ShaderNodeSocket &socket,
                                                                  const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<ScaleAndOffsetFloatShaderNode>(socket.nodeDescIndex);
        float value = calcNode(nodeData.nodeValue, 0.0f, surfPt, wls);
        float scale = calcNode(nodeData.nodeScale, nodeData.immScale, surfPt, wls);
        float offset = calcNode(nodeData.nodeOffset, nodeData.immOffset, surfPt, wls);
        return scale * value + offset;
    }



    RT_CALLABLE_PROGRAM SampledSpectrum TripletSpectrumShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                           const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<TripletSpectrumShaderNode>(socket.nodeDescIndex);
        return nodeData.value.evaluate(wls);
    }



#if defined(VLR_USE_SPECTRAL_RENDERING)
    template <>
    RT_FUNCTION RegularSampledSpectrumShaderNode* getData<RegularSampledSpectrumShaderNode>(uint32_t nodeDescIndex) {
        return pv_largeNodeDescriptorBuffer[nodeDescIndex].getData<RegularSampledSpectrumShaderNode>();
    }
#endif
    
    RT_CALLABLE_PROGRAM SampledSpectrum RegularSampledSpectrumShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                                  const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<RegularSampledSpectrumShaderNode>(socket.nodeDescIndex);
#if defined(VLR_USE_SPECTRAL_RENDERING)
        return RegularSampledSpectrum(nodeData.minLambda, nodeData.maxLambda, nodeData.values, nodeData.numSamples).evaluate(wls);
#else
        return nodeData.value.evaluate(wls);
#endif
    }



#if defined(VLR_USE_SPECTRAL_RENDERING)
    template <>
    RT_FUNCTION IrregularSampledSpectrumShaderNode* getData<IrregularSampledSpectrumShaderNode>(uint32_t nodeDescIndex) {
        return pv_largeNodeDescriptorBuffer[nodeDescIndex].getData<IrregularSampledSpectrumShaderNode>();
    }
#endif

    RT_CALLABLE_PROGRAM SampledSpectrum IrregularSampledSpectrumShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                                    const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<IrregularSampledSpectrumShaderNode>(socket.nodeDescIndex);
#if defined(VLR_USE_SPECTRAL_RENDERING)
        return IrregularSampledSpectrum(nodeData.lambdas, nodeData.values, nodeData.numSamples).evaluate(wls);
#else
        return nodeData.value.evaluate(wls);
#endif
    }



    template <>
    RT_FUNCTION Float3ToSpectrumShaderNode* getData<Float3ToSpectrumShaderNode>(uint32_t nodeDescIndex) {
        return pv_mediumNodeDescriptorBuffer[nodeDescIndex].getData<Float3ToSpectrumShaderNode>();
    }
    
    RT_CALLABLE_PROGRAM SampledSpectrum Float3ToSpectrumShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                            const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float3ToSpectrumShaderNode>(socket.nodeDescIndex);
        auto defaultValue = optix::make_float3(nodeData.immFloat3[0], nodeData.immFloat3[1], nodeData.immFloat3[2]);
        optix::float3 f3Value = calcNode(nodeData.nodeFloat3, defaultValue, surfPt, wls);
#if defined(VLR_USE_SPECTRAL_RENDERING)
        return UpsampledSpectrum(nodeData.spectrumType, nodeData.colorSpace,
                                 clamp(0.5f * f3Value.x + 0.5f, 0.0f, 1.0f),
                                 clamp(0.5f * f3Value.y + 0.5f, 0.0f, 1.0f),
                                 clamp(0.5f * f3Value.z + 0.5f, 0.0f, 1.0f)).evaluate(wls);
#else
        return SampledSpectrum(clamp(0.5f * f3Value.x + 0.5f, 0.0f, 1.0f),
                               clamp(0.5f * f3Value.y + 0.5f, 0.0f, 1.0f),
                               clamp(0.5f * f3Value.z + 0.5f, 0.0f, 1.0f));
#endif
    }



    RT_CALLABLE_PROGRAM Point3D ScaleAndOffsetUVTextureMap2DShaderNode_TextureCoordinates(const ShaderNodeSocket &socket,
                                                                                          const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<ScaleAndOffsetUVTextureMap2DShaderNode>(socket.nodeDescIndex);
        return Point3D(nodeData.scale[0] * surfPt.texCoord.u + nodeData.offset[0],
                       nodeData.scale[1] * surfPt.texCoord.v + nodeData.offset[1],
                       0.0f);
    }



    RT_CALLABLE_PROGRAM float Image2DTextureShaderNode_float(const ShaderNodeSocket &socket,
                                                             const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        if (socket.option == 0)
            return texValue.x;
        else if (socket.option == 1)
            return texValue.y;
        else if (socket.option == 2)
            return texValue.z;
        else if (socket.option == 3)
            return texValue.w;

        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Image2DTextureShaderNode_float2(const ShaderNodeSocket &socket,
                                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        if (socket.option == 0)
            return optix::make_float2(texValue.x, texValue.y);
        else if (socket.option == 1)
            return optix::make_float2(texValue.y, texValue.z);
        else if (socket.option == 2)
            return optix::make_float2(texValue.z, texValue.w);

        return optix::make_float2(0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float3 Image2DTextureShaderNode_float3(const ShaderNodeSocket &socket,
                                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        if (socket.option == 0)
            return optix::make_float3(texValue.x, texValue.y, texValue.z);
        else if (socket.option == 1)
            return optix::make_float3(texValue.y, texValue.z, texValue.w);

        return optix::make_float3(0.0f, 0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float4 Image2DTextureShaderNode_float4(const ShaderNodeSocket &socket,
                                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        return texValue;
    }

    RT_CALLABLE_PROGRAM Normal3D Image2DTextureShaderNode_Normal3D(const ShaderNodeSocket &socket,
                                                                   const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        if (socket.option == 0)
            return 2 * Normal3D(texValue.x, texValue.y, texValue.z) - 1.0f;
        else if (socket.option == 1)
            return 2 * Normal3D(texValue.y, texValue.z, texValue.w) - 1.0f;

        return Normal3D(0.0f, 0.0f, 1.0f);
    }

    RT_CALLABLE_PROGRAM SampledSpectrum Image2DTextureShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                          const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);
        DataFormat dataFormat = nodeData.getDataFormat();
        if (dataFormat == DataFormat::Gray32F ||
            dataFormat == DataFormat::Gray8 ||
            dataFormat == DataFormat::GrayA8x2)
            texValue.z = texValue.y = texValue.x;

#if defined(VLR_USE_SPECTRAL_RENDERING)
        UpsampledSpectrum spectrum;
        if (dataFormat == DataFormat::uvsA8x4 ||
            dataFormat == DataFormat::uvsA16Fx4) {
            float u = texValue.x;
            float v = texValue.y;
            float s = texValue.z;
            if (dataFormat == DataFormat::uvsA8x4) {
                u *= UpsampledSpectrum::GridWidth();
                v *= UpsampledSpectrum::GridHeight();
                s *= 3;
            }
            // JP: uvsA16Fの場合もInf回避のために EqualEnergyReflectance で割っていないので
            //     どちらのフォーマットだとしても割る。
            // EN: 
            s /= UpsampledSpectrum::EqualEnergyReflectance();
            spectrum = UpsampledSpectrum(u, v, s);
        }
        else {
            spectrum = UpsampledSpectrum(nodeData.getSpectrumType(), nodeData.getColorSpace(), texValue.x, texValue.y, texValue.z);
        }
        return spectrum.evaluate(wls);
#else
        return SampledSpectrum(texValue.x, texValue.y, texValue.z); // assume given data is in rendering RGB.
#endif
    }

    RT_CALLABLE_PROGRAM float Image2DTextureShaderNode_Alpha(const ShaderNodeSocket &socket,
                                                             const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        if (socket.option == 0)
            return texValue.x;
        else if (socket.option == 1)
            return texValue.y;
        else if (socket.option == 2)
            return texValue.z;
        else if (socket.option == 3)
            return texValue.w;

        return 0.0f;
    }



    RT_CALLABLE_PROGRAM SampledSpectrum EnvironmentTextureShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<EnvironmentTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

#if defined(VLR_USE_SPECTRAL_RENDERING)
        DataFormat dataFormat = nodeData.getDataFormat();

        UpsampledSpectrum spectrum;
        if (dataFormat == DataFormat::uvsA16Fx4) {
            float u = texValue.x;
            float v = texValue.y;
            float s = texValue.z;
            s /= UpsampledSpectrum::EqualEnergyReflectance();
            spectrum = UpsampledSpectrum(u, v, s);
        }
        else {
            spectrum = UpsampledSpectrum(SpectrumType::LightSource, nodeData.getColorSpace(), texValue.x, texValue.y, texValue.z);
        }
        return spectrum.evaluate(wls);
#else
        return SampledSpectrum(texValue.x, texValue.y, texValue.z); // assume given data is in rendering RGB.
#endif
    }
}
