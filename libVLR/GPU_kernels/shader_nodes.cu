#include "hip/hip_runtime.h"
﻿#include "kernel_common.cuh"

namespace VLR {
    template <typename T>
    RT_FUNCTION T* getData(uint32_t nodeDescIndex) {
        constexpr uint32_t sizeOfNodeInDW = sizeof(T) / 4;
        if /*constexpr*/ (sizeOfNodeInDW <= VLR_MAX_NUM_SMALL_NODE_DESCRIPTOR_SLOTS)
            return pv_smallNodeDescriptorBuffer[nodeDescIndex].getData<T>();
        else if /*constexpr*/ (sizeOfNodeInDW <= VLR_MAX_NUM_MEDIUM_NODE_DESCRIPTOR_SLOTS)
            return pv_mediumNodeDescriptorBuffer[nodeDescIndex].getData<T>();
        else if /*constexpr*/ (sizeOfNodeInDW <= VLR_MAX_NUM_LARGE_NODE_DESCRIPTOR_SLOTS)
            return pv_largeNodeDescriptorBuffer[nodeDescIndex].getData<T>();
        return nullptr;
    }



    RT_CALLABLE_PROGRAM Point3D GeometryShaderNode_Point3D(const ShaderNodeSocket &socket,
                                                           const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        return surfPt.position;
    }

    RT_CALLABLE_PROGRAM Normal3D GeometryShaderNode_Normal3D(const ShaderNodeSocket &socket,
                                                             const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        if (socket.option == 0)
            return surfPt.geometricNormal;
        else if (socket.option == 1)
            return surfPt.shadingFrame.z;
        return Normal3D(0, 0, 0);
    }

    RT_CALLABLE_PROGRAM Vector3D GeometryShaderNode_Vector3D(const ShaderNodeSocket &socket,
                                                             const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        if (socket.option == 0)
            return surfPt.shadingFrame.x;
        else if (socket.option == 1)
            return surfPt.shadingFrame.y;
        return Vector3D::Zero();
    }

    RT_CALLABLE_PROGRAM Point3D GeometryShaderNode_TextureCoordinates(const ShaderNodeSocket &socket,
                                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        return Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0);
    }



    RT_CALLABLE_PROGRAM float Float2ShaderNode_float1(const ShaderNodeSocket &socket,
                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float2ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return calcNode(nodeData.node0, nodeData.imm0, surfPt, wls);
        else if (socket.option == 1)
            return calcNode(nodeData.node1, nodeData.imm1, surfPt, wls);
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Float2ShaderNode_float2(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float2ShaderNode>(socket.nodeDescIndex);
        return optix::make_float2(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                  calcNode(nodeData.node1, nodeData.imm1, surfPt, wls));
    }



    RT_CALLABLE_PROGRAM float Float3ShaderNode_float1(const ShaderNodeSocket &socket,
                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float3ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return calcNode(nodeData.node0, nodeData.imm0, surfPt, wls);
        else if (socket.option == 1)
            return calcNode(nodeData.node1, nodeData.imm1, surfPt, wls);
        else if (socket.option == 2)
            return calcNode(nodeData.node2, nodeData.imm2, surfPt, wls);
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Float3ShaderNode_float2(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float3ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return optix::make_float2(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                      calcNode(nodeData.node1, nodeData.imm1, surfPt, wls));
        else if (socket.option == 1)
            return optix::make_float2(calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                      calcNode(nodeData.node2, nodeData.imm2, surfPt, wls));
        return optix::make_float2(0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float3 Float3ShaderNode_float3(const ShaderNodeSocket &socket, 
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float3ShaderNode>(socket.nodeDescIndex);
        return optix::make_float3(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                  calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                  calcNode(nodeData.node2, nodeData.imm2, surfPt, wls));
    }



    RT_CALLABLE_PROGRAM float Float4ShaderNode_float1(const ShaderNodeSocket &socket,
                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float4ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return calcNode(nodeData.node0, nodeData.imm0, surfPt, wls);
        else if (socket.option == 1)
            return calcNode(nodeData.node1, nodeData.imm1, surfPt, wls);
        else if (socket.option == 2)
            return calcNode(nodeData.node2, nodeData.imm2, surfPt, wls);
        else if (socket.option == 3)
            return calcNode(nodeData.node3, nodeData.imm3, surfPt, wls);
        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Float4ShaderNode_float2(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float4ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return optix::make_float2(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                      calcNode(nodeData.node1, nodeData.imm1, surfPt, wls));
        else if (socket.option == 1)
            return optix::make_float2(calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                      calcNode(nodeData.node2, nodeData.imm2, surfPt, wls));
        else if (socket.option == 2)
            return optix::make_float2(calcNode(nodeData.node2, nodeData.imm2, surfPt, wls),
                                      calcNode(nodeData.node3, nodeData.imm3, surfPt, wls));
        return optix::make_float2(0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float3 Float4ShaderNode_float3(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float4ShaderNode>(socket.nodeDescIndex);
        if (socket.option == 0)
            return optix::make_float3(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                      calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                      calcNode(nodeData.node2, nodeData.imm2, surfPt, wls));
        else if (socket.option == 1)
            return optix::make_float3(calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                      calcNode(nodeData.node2, nodeData.imm2, surfPt, wls),
                                      calcNode(nodeData.node3, nodeData.imm3, surfPt, wls));
        return optix::make_float3(0.0f, 0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float4 Float4ShaderNode_float4(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float4ShaderNode>(socket.nodeDescIndex);
        return optix::make_float4(calcNode(nodeData.node0, nodeData.imm0, surfPt, wls),
                                  calcNode(nodeData.node1, nodeData.imm1, surfPt, wls),
                                  calcNode(nodeData.node2, nodeData.imm2, surfPt, wls),
                                  calcNode(nodeData.node3, nodeData.imm3, surfPt, wls));
    }



    RT_CALLABLE_PROGRAM float ScaleAndOffsetFloatShaderNode_float1(const ShaderNodeSocket &socket,
                                                                   const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<ScaleAndOffsetFloatShaderNode>(socket.nodeDescIndex);
        float value = calcNode(nodeData.nodeValue, 0.0f, surfPt, wls);
        float scale = calcNode(nodeData.nodeScale, nodeData.immScale, surfPt, wls);
        float offset = calcNode(nodeData.nodeOffset, nodeData.immOffset, surfPt, wls);
        return scale * value + offset;
    }



    RT_CALLABLE_PROGRAM SampledSpectrum TripletSpectrumShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                           const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<TripletSpectrumShaderNode>(socket.nodeDescIndex);
        return nodeData.value.evaluate(wls);
    }



    RT_CALLABLE_PROGRAM SampledSpectrum RegularSampledSpectrumShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                                  const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<RegularSampledSpectrumShaderNode>(socket.nodeDescIndex);
#if defined(VLR_USE_SPECTRAL_RENDERING)
        return RegularSampledSpectrum(nodeData.minLambda, nodeData.maxLambda, nodeData.values, nodeData.numSamples).evaluate(wls);
#else
        return nodeData.value.evaluate(wls);
#endif
    }



    RT_CALLABLE_PROGRAM SampledSpectrum IrregularSampledSpectrumShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                                    const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<IrregularSampledSpectrumShaderNode>(socket.nodeDescIndex);
#if defined(VLR_USE_SPECTRAL_RENDERING)
        return IrregularSampledSpectrum(nodeData.lambdas, nodeData.values, nodeData.numSamples).evaluate(wls);
#else
        return nodeData.value.evaluate(wls);
#endif
    }



    RT_CALLABLE_PROGRAM SampledSpectrum Float3ToSpectrumShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                            const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Float3ToSpectrumShaderNode>(socket.nodeDescIndex);
        auto defaultValue = optix::make_float3(nodeData.immFloat3[0], nodeData.immFloat3[1], nodeData.immFloat3[2]);
        optix::float3 f3Value = calcNode(nodeData.nodeFloat3, defaultValue, surfPt, wls);
#if defined(VLR_USE_SPECTRAL_RENDERING)
        return UpsampledSpectrum(nodeData.spectrumType, nodeData.colorSpace,
                                 clamp(0.5f * f3Value.x + 0.5f, 0.0f, 1.0f),
                                 clamp(0.5f * f3Value.y + 0.5f, 0.0f, 1.0f),
                                 clamp(0.5f * f3Value.z + 0.5f, 0.0f, 1.0f)).evaluate(wls);
#else
        return SampledSpectrum(clamp(0.5f * f3Value.x + 0.5f, 0.0f, 1.0f),
                               clamp(0.5f * f3Value.y + 0.5f, 0.0f, 1.0f),
                               clamp(0.5f * f3Value.z + 0.5f, 0.0f, 1.0f));
#endif
    }



    RT_CALLABLE_PROGRAM Point3D ScaleAndOffsetUVTextureMap2DShaderNode_TextureCoordinates(const ShaderNodeSocket &socket,
                                                                                          const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<ScaleAndOffsetUVTextureMap2DShaderNode>(socket.nodeDescIndex);
        return Point3D(nodeData.scale[0] * surfPt.texCoord.u + nodeData.offset[0],
                       nodeData.scale[1] * surfPt.texCoord.v + nodeData.offset[1],
                       0.0f);
    }



    RT_CALLABLE_PROGRAM float Image2DTextureShaderNode_float1(const ShaderNodeSocket &socket,
                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        if (socket.option == 0)
            return texValue.x;
        else if (socket.option == 1)
            return texValue.y;
        else if (socket.option == 2)
            return texValue.z;
        else if (socket.option == 3)
            return texValue.w;

        return 0.0f;
    }

    RT_CALLABLE_PROGRAM optix::float2 Image2DTextureShaderNode_float2(const ShaderNodeSocket &socket,
                                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        if (socket.option == 0)
            return optix::make_float2(texValue.x, texValue.y);
        else if (socket.option == 1)
            return optix::make_float2(texValue.y, texValue.z);
        else if (socket.option == 2)
            return optix::make_float2(texValue.z, texValue.w);

        return optix::make_float2(0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float3 Image2DTextureShaderNode_float3(const ShaderNodeSocket &socket,
                                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        if (socket.option == 0)
            return optix::make_float3(texValue.x, texValue.y, texValue.z);
        else if (socket.option == 1)
            return optix::make_float3(texValue.y, texValue.z, texValue.w);

        return optix::make_float3(0.0f, 0.0f, 0.0f);
    }

    RT_CALLABLE_PROGRAM optix::float4 Image2DTextureShaderNode_float4(const ShaderNodeSocket &socket,
                                                                      const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        return texValue;
    }

    RT_CALLABLE_PROGRAM Normal3D Image2DTextureShaderNode_Normal3D(const ShaderNodeSocket &socket,
                                                                   const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);
        BumpType bumpType = nodeData.getBumpType();

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue;
        if (bumpType != BumpType::HeightMap) {
            texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);
        }
        else {
            // w z
            // x y
            texValue = optix::rtTex2DGather<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, socket.option);
        }

        Normal3D ret(0.0f, 0.0f, 1.0f);
        if (bumpType != BumpType::HeightMap && socket.option < 2) {
            if (socket.option == 0)
                ret = Normal3D(texValue.x, texValue.y, texValue.z);
            else if (socket.option == 1)
                ret = Normal3D(texValue.y, texValue.z, texValue.w);

            ret = 2 * ret - 1.0f;

            if (bumpType == BumpType::NormalMap_DirectX)
                ret.y *= -1;
        }
        else if (bumpType == BumpType::HeightMap) {
            const float coeff = 5.0f;
            float dhdu = coeff * (texValue.y - texValue.x);
            float dhdv = coeff * (texValue.x - texValue.w);
            // cross(Vector3D(0, -1, dhdv), 
            //       Vector3D(1,  0, dhdu))
            ret = Normal3D(-dhdu, dhdv, 1);
        }

        return normalize(ret);
    }

    RT_CALLABLE_PROGRAM SampledSpectrum Image2DTextureShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                          const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);
        DataFormat dataFormat = nodeData.getDataFormat();
        if (dataFormat == DataFormat::Gray32F ||
            dataFormat == DataFormat::Gray8 ||
            dataFormat == DataFormat::GrayA8x2)
            texValue.z = texValue.y = texValue.x;

#if defined(VLR_USE_SPECTRAL_RENDERING)
        UpsampledSpectrum spectrum;
        if (dataFormat == DataFormat::uvsA8x4 ||
            dataFormat == DataFormat::uvsA16Fx4) {
            float u = texValue.x;
            float v = texValue.y;
            float s = texValue.z;
            if (dataFormat == DataFormat::uvsA8x4) {
                u *= UpsampledSpectrum::GridWidth();
                v *= UpsampledSpectrum::GridHeight();
                s *= 3;
            }
            // JP: uvsA16Fの場合もInf回避のために EqualEnergyReflectance で割っていないので
            //     どちらのフォーマットだとしても割る。
            // EN: 
            s /= UpsampledSpectrum::EqualEnergyReflectance();
            spectrum = UpsampledSpectrum(u, v, s);
        }
        else {
            spectrum = UpsampledSpectrum(nodeData.getSpectrumType(), nodeData.getColorSpace(), texValue.x, texValue.y, texValue.z);
        }
        return spectrum.evaluate(wls);
#else
        return SampledSpectrum(texValue.x, texValue.y, texValue.z); // assume given data is in rendering RGB.
#endif
    }

    RT_CALLABLE_PROGRAM float Image2DTextureShaderNode_Alpha(const ShaderNodeSocket &socket,
                                                             const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<Image2DTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

        if (socket.option == 0)
            return texValue.x;
        else if (socket.option == 1)
            return texValue.y;
        else if (socket.option == 2)
            return texValue.z;
        else if (socket.option == 3)
            return texValue.w;

        return 0.0f;
    }



    RT_CALLABLE_PROGRAM SampledSpectrum EnvironmentTextureShaderNode_Spectrum(const ShaderNodeSocket &socket,
                                                                              const SurfacePoint &surfPt, const WavelengthSamples &wls) {
        auto &nodeData = *getData<EnvironmentTextureShaderNode>(socket.nodeDescIndex);

        Point3D texCoord = calcNode(nodeData.nodeTexCoord, Point3D(surfPt.texCoord.u, surfPt.texCoord.v, 0.0f), surfPt, wls);
        optix::float4 texValue = optix::rtTex2DLod<optix::float4>(nodeData.textureID, texCoord.x, texCoord.y, 0.0f);

#if defined(VLR_USE_SPECTRAL_RENDERING)
        DataFormat dataFormat = nodeData.getDataFormat();

        UpsampledSpectrum spectrum;
        if (dataFormat == DataFormat::uvsA16Fx4) {
            float u = texValue.x;
            float v = texValue.y;
            float s = texValue.z;
            s /= UpsampledSpectrum::EqualEnergyReflectance();
            spectrum = UpsampledSpectrum(u, v, s);
        }
        else {
            spectrum = UpsampledSpectrum(SpectrumType::LightSource, nodeData.getColorSpace(), texValue.x, texValue.y, texValue.z);
        }
        return spectrum.evaluate(wls);
#else
        return SampledSpectrum(texValue.x, texValue.y, texValue.z); // assume given data is in rendering RGB.
#endif
    }
}
