#include "hip/hip_runtime.h"
﻿#include "kernel_common.cuh"

namespace VLR {
    // per GeometryInstance
    // closestHitProgramなどから呼ばれるdecodeHitPoint等で読み出すためにはGeometryInstanceレベルにバインドする必要がある。
    rtBuffer<Vertex> pv_vertexBuffer;
    rtBuffer<Triangle> pv_triangleBuffer;
    rtDeclareVariable(float, pv_sumImportances, , );

    // Intersection Program
    RT_PROGRAM void intersectTriangle(int32_t primIdx) {
        const Triangle &triangle = pv_triangleBuffer[primIdx];
        const Vertex &v0 = pv_vertexBuffer[triangle.index0];
        const Vertex &v1 = pv_vertexBuffer[triangle.index1];
        const Vertex &v2 = pv_vertexBuffer[triangle.index2];

        // use a triangle intersection function defined in optix_math_namespace.h
        optix::float3 gn;
        float t;
        float b0, b1, b2;
        if (!intersect_triangle(sm_ray, asOptiXType(v0.position), asOptiXType(v1.position), asOptiXType(v2.position),
                                gn, t, b1, b2))
            return;

        if (!rtPotentialIntersection(t))
            return;

        b0 = 1.0f - b1 - b2;
        a_hitPointParam.b0 = b0;
        a_hitPointParam.b1 = b1;
        a_hitPointParam.primIndex = primIdx;

        const uint32_t materialIndex = 0;
        rtReportIntersection(materialIndex);
    }

    // Bounding Box Program
    RT_PROGRAM void calcBBoxForTriangle(int32_t primIdx, float result[6]) {
        const Triangle &triangle = pv_triangleBuffer[primIdx];
        const Point3D &p0 = pv_vertexBuffer[triangle.index0].position;
        const Point3D &p1 = pv_vertexBuffer[triangle.index1].position;
        const Point3D &p2 = pv_vertexBuffer[triangle.index2].position;

        //optix::Aabb* bbox = (optix::Aabb*)result;
        //*bbox = optix::Aabb(asOptiXType(p0), asOptiXType(p1), asOptiXType(p2));

        BoundingBox3D* bbox = (BoundingBox3D*)result;
        *bbox = BoundingBox3D(Point3D(INFINITY), Point3D(-INFINITY));
        bbox->unify(p0);
        bbox->unify(p1);
        bbox->unify(p2);
    }

    // Attribute Program (for GeometryTriangles)
    RT_PROGRAM void calcAttributeForTriangle() {
        optix::float2 bc = rtGetTriangleBarycentrics();
        a_hitPointParam.b0 = 1 - bc.x - bc.y;
        a_hitPointParam.b1 = bc.x;
        a_hitPointParam.primIndex = rtGetPrimitiveIndex();
    }



    // bound
    RT_CALLABLE_PROGRAM void decodeHitPointForTriangle(const HitPointParameter &param, SurfacePoint* surfPt, float* hypAreaPDF) {
        const Triangle &triangle = pv_triangleBuffer[param.primIndex];
        const Vertex &v0 = pv_vertexBuffer[triangle.index0];
        const Vertex &v1 = pv_vertexBuffer[triangle.index1];
        const Vertex &v2 = pv_vertexBuffer[triangle.index2];

        Vector3D e1 = transform(RT_OBJECT_TO_WORLD, v1.position - v0.position);
        Vector3D e2 = transform(RT_OBJECT_TO_WORLD, v2.position - v0.position);
        Normal3D geometricNormal = cross(e1, e2);
        float area = geometricNormal.length() / 2; // TODO: スケーリングの考慮。
        geometricNormal /= 2 * area;

        // JP: プログラムがこの点を光源としてサンプルする場合の面積に関する(仮想的な)PDFを求める。
        // EN: calculate a hypothetical area PDF value in the case where the program sample this point as light.
        float probLightPrim = area / pv_sumImportances;
        *hypAreaPDF = probLightPrim / area;

        float b0 = param.b0, b1 = param.b1, b2 = 1.0f - param.b0 - param.b1;
        Point3D position = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        Normal3D shadingNormal = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        Vector3D tc0Direction = b0 * v0.tc0Direction + b1 * v1.tc0Direction + b2 * v2.tc0Direction;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        position = transform(RT_OBJECT_TO_WORLD, position);
        shadingNormal = normalize(transform(RT_OBJECT_TO_WORLD, shadingNormal));
        tc0Direction = transform(RT_OBJECT_TO_WORLD, tc0Direction);

        // JP: 法線と接線が直交することを保証する。
        //     直交性の消失は重心座標補間によっておこる？
        // EN: guarantee the orthogonality between the normal and tangent.
        //     Orthogonality break might be caused by barycentric interpolation?
        float dotNT = dot(shadingNormal, tc0Direction);
        tc0Direction = normalize(tc0Direction - dotNT * shadingNormal);

        surfPt->position = position;
        surfPt->shadingFrame = ReferenceFrame(tc0Direction, shadingNormal);
        surfPt->isPoint = false;
        surfPt->atInfinity = false;
        surfPt->geometricNormal = geometricNormal;
        surfPt->u = b0;
        surfPt->v = b1;
        surfPt->texCoord = texCoord;
    }



    RT_CALLABLE_PROGRAM void sampleTriangleMesh(const GeometryInstanceDescriptor::Body &desc, const SurfaceLightPosSample &sample, SurfaceLightPosQueryResult* result) {
        float primProb;
        uint32_t primIdx = desc.asTriMesh.primDistribution.sample(sample.uElem, &primProb);

        const Triangle &triangle = desc.asTriMesh.triangleBuffer[primIdx];
        const Vertex &v0 = desc.asTriMesh.vertexBuffer[triangle.index0];
        const Vertex &v1 = desc.asTriMesh.vertexBuffer[triangle.index1];
        const Vertex &v2 = desc.asTriMesh.vertexBuffer[triangle.index2];

        StaticTransform transform = desc.asTriMesh.transform;

        Vector3D e1 = transform * (v1.position - v0.position);
        Vector3D e2 = transform * (v2.position - v0.position);
        Normal3D geometricNormal = cross(e1, e2);
        float area = geometricNormal.length() / 2;
        geometricNormal /= 2 * area;

        result->areaPDF = primProb / area;
        result->posType = DirectionType::Emission() | DirectionType::LowFreq();

        float b0, b1, b2;
        uniformSampleTriangle(sample.uPos[0], sample.uPos[1], &b0, &b1);
        b2 = 1.0f - b0 - b1;

        Point3D position = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        Normal3D shadingNormal = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        Vector3D tc0Direction = b0 * v0.tc0Direction + b1 * v1.tc0Direction + b2 * v2.tc0Direction;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        position = transform * position;
        shadingNormal = normalize(transform * shadingNormal);
        tc0Direction = transform * tc0Direction;

        // JP: 法線と接線が直交することを保証する。
        //     直交性の消失は重心座標補間によっておこる？
        // EN: guarantee the orthogonality between the normal and tangent.
        //     Orthogonality break might be caused by barycentric interpolation?
        float dotNT = dot(shadingNormal, tc0Direction);
        tc0Direction = normalize(tc0Direction - dotNT * shadingNormal);

        SurfacePoint &surfPt = result->surfPt;

        surfPt.position = position;
        surfPt.shadingFrame = ReferenceFrame(tc0Direction, shadingNormal);
        surfPt.isPoint = false;
        surfPt.atInfinity = false;
        surfPt.geometricNormal = geometricNormal;
        surfPt.u = b0;
        surfPt.v = b1;
        surfPt.texCoord = texCoord;
    }
}
