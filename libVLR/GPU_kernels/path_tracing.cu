#include "hip/hip_runtime.h"
﻿#include "cameras.cu"
#include "light_transport_common.cuh"

namespace VLR {
    // Context-scope Variables
    rtDeclareVariable(optix::uint2, pv_imageSize, , );
    rtDeclareVariable(uint32_t, pv_numAccumFrames, , );
    rtDeclareVariable(ProgSigSampleLensPosition, pv_progSampleLensPosition, , );
    rtDeclareVariable(ProgSigSampleIDF, pv_progSampleIDF, , );
    rtBuffer<KernelRNG, 2> pv_rngBuffer;
    rtBuffer<SpectrumStorage, 2> pv_outputBuffer;
    //rtBuffer<RGBSpectrum, 2> pv_outputNormalBuffer;
    //rtBuffer<RGBSpectrum, 2> pv_outputAlbedoBuffer;

	

	#define CLAMPINTENSITY(c, clampValue)		const float v=std::fmax(c.r,std::fmax(c.g, c.b)); \
							if(v>clampValue){const float m=clampValue/v;c.r*=m; \
							c.g*=m;c.b*=m; /* don't touch w */ }

	
    // Common Closest Hit Program for All Primitive Types and Materials
    RT_PROGRAM void pathTracingIteration() {
		// first hit is an object
        if (sm_payload.contribution.a == -1.0f)
            sm_payload.contribution.a = 1.f;

        KernelRNG &rng = sm_payload.rng;
        WavelengthSamples &wls = sm_payload.wls;

        SurfacePoint surfPt;
        float hypAreaPDF;
        calcSurfacePoint(&surfPt, &hypAreaPDF);

        const SurfaceMaterialDescriptor matDesc = pv_materialDescriptorBuffer[pv_materialIndex];
        BSDF bsdf(matDesc, surfPt, wls);
        EDF edf(matDesc, surfPt, wls);

        Vector3D dirOutLocal = surfPt.shadingFrame.toLocal(-asVector3D(sm_ray.direction));

        // implicit light sampling
        SampledSpectrum spEmittance = edf.evaluateEmittance();
        if (spEmittance.hasNonZero()) {
            SampledSpectrum Le = spEmittance * edf.evaluate(EDFQuery(), dirOutLocal);

            float MISWeight = 1.0f;
            if (!sm_payload.prevSampledType.isDelta() && sm_ray.ray_type != RayType::Primary) {
                float bsdfPDF = sm_payload.prevDirPDF;
                float dist2 = surfPt.calcSquaredDistance(asPoint3D(sm_ray.origin));
                float lightPDF = pv_importance / getSumLightImportances() * hypAreaPDF * dist2 / std::fabs(dirOutLocal.z);
                MISWeight = (bsdfPDF * bsdfPDF) / (lightPDF * lightPDF + bsdfPDF * bsdfPDF);
            }

            sm_payload.contribution += sm_payload.alpha * Le * MISWeight;
			CLAMPINTENSITY(sm_payload.contribution, 10.f);
        }
        if (surfPt.atInfinity || sm_payload.maxLengthTerminate)
            return;

        // Russian roulette
        float continueProb = std::fmin(sm_payload.alpha.importance(wls.selectedLambdaIndex()) / sm_payload.initImportance, 1.0f);
        if (rng.getFloat0cTo1o() >= continueProb)
            return;
        sm_payload.alpha /= continueProb;

        Normal3D geomNormalLocal = surfPt.shadingFrame.toLocal(surfPt.geometricNormal);
        BSDFQuery fsQuery(dirOutLocal, geomNormalLocal, DirectionType::All(), wls);

        // get base color for denoiser
   /*     if (sm_payload.contribution.a == -1.0f) {
            sm_payload.contribution.a = 1.f;

      /*      const BSDFProcedureSet procSet = pv_bsdfProcedureSetBuffer[matDesc.bsdfProcedureSetIndex];
            auto progGetBaseColor = (ProgSigBSDFGetBaseColor)procSet.progGetBaseColor;
            sm_payload.albedo = progGetBaseColor((const uint32_t *)&bsdf);
            //	sm_payload.normal = RGBSpectrum(surfPt.geometricNormal.x, surfPt.geometricNormal.y, surfPt.geometricNormal.z);
            //	sm_payload.normal = RGBSpectrum(geomNormalLocal.x, geomNormalLocal.y, geomNormalLocal.z);
    /*		auto rotMat = Matrix4x4(pv_perspectiveCamera.orientation.toMatrix3x3());
            //rotMat = rotateY(1.57f) * rotMat;
            rotMat = translate(pv_perspectiveCamera.position.x, pv_perspectiveCamera.position.y, pv_perspectiveCamera.position.z) * rotMat;
            rotMat = invert(rotMat);
            auto normalCam = normalize(rotMat * surfPt.geometricNormal);
            sm_payload.normal = RGBSpectrum(-normalCam.x, normalCam.y, -normalCam.z);
    *///	}

        // Next Event Estimation (explicit light sampling)
        if (bsdf.hasNonDelta()) {
            SurfaceLight light;
            float lightProb;
            float uPrim;
            selectSurfaceLight(rng.getFloat0cTo1o(), &light, &lightProb, &uPrim);

            SurfaceLightPosSample lpSample(uPrim, rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
            SurfaceLightPosQueryResult lpResult;
            light.sample(lpSample, &lpResult);

            const SurfaceMaterialDescriptor lightMatDesc = pv_materialDescriptorBuffer[lpResult.materialIndex];
            EDF ledf(lightMatDesc, lpResult.surfPt, wls);
            SampledSpectrum M = ledf.evaluateEmittance();

            Vector3D shadowRayDir;
            float squaredDistance;
            float fractionalVisibility;
			SampledSpectrum shadow_color;
            if (M.hasNonZero() && testVisibility(surfPt, lpResult.surfPt, &shadowRayDir, &squaredDistance, &fractionalVisibility, &shadow_color)) {
                Vector3D shadowRayDir_l = lpResult.surfPt.toLocal(-shadowRayDir);
                Vector3D shadowRayDir_sn = surfPt.toLocal(shadowRayDir);

				SampledSpectrum Le = M * ledf.evaluate(EDFQuery(), shadowRayDir_l) *shadow_color;
                float lightPDF = lightProb * lpResult.areaPDF;

                SampledSpectrum fs = bsdf.evaluate(fsQuery, shadowRayDir_sn);
                float cosLight = lpResult.surfPt.calcCosTerm(-shadowRayDir);
                float bsdfPDF = bsdf.evaluatePDF(fsQuery, shadowRayDir_sn) * cosLight / squaredDistance;

                float MISWeight = 1.0f;
                if (!lpResult.posType.isDelta() && !std::isinf(lightPDF))
                    MISWeight = (lightPDF * lightPDF) / (lightPDF * lightPDF + bsdfPDF * bsdfPDF);

                float G = fractionalVisibility * absDot(shadowRayDir_sn, geomNormalLocal) * cosLight / squaredDistance;
                float scalarCoeff = G * MISWeight / lightPDF; // 直接contributionの計算式に入れるとCUDAのバグなのかおかしな結果になる。
                sm_payload.contribution += sm_payload.alpha * Le * fs * scalarCoeff;
				CLAMPINTENSITY(sm_payload.contribution, 100.f);
            }
        }

        BSDFSample sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        BSDFQueryResult fsResult;
        SampledSpectrum fs = bsdf.sample(fsQuery, sample, &fsResult);
        if (fs == SampledSpectrum::Zero() || fsResult.dirPDF == 0.0f)
            return;
        if (fsResult.sampledType.isDispersive() && !wls.singleIsSelected()) {
            fsResult.dirPDF /= SampledSpectrum::NumComponents();
            wls.setSingleIsSelected();
        }

        float cosFactor = dot(fsResult.dirLocal, geomNormalLocal);
        sm_payload.alpha *= fs * (std::fabs(cosFactor) / fsResult.dirPDF);
	//	CLAMPINTENSITY(sm_payload.alpha);

        Vector3D dirIn = surfPt.fromLocal(fsResult.dirLocal);
        sm_payload.origin = offsetRayOrigin(surfPt.position, cosFactor > 0.0f ? surfPt.geometricNormal : -surfPt.geometricNormal);
        sm_payload.direction = dirIn;
        sm_payload.prevDirPDF = fsResult.dirPDF;
        sm_payload.prevSampledType = fsResult.sampledType;
        sm_payload.terminate = false;
    }



    // JP: 本当は無限大の球のIntersection/Bounding Box Programを使用して環境光に関する処理もClosest Hit Programで統一的に行いたい。
    //     が、OptiXのBVHビルダーがLBVHベースなので無限大のAABBを生成するのは危険。
    //     仕方なくMiss Programで環境光を処理する。
    RT_PROGRAM void pathTracingMiss() {
        // first hit is the background, set alpha to °
        if (sm_payload.contribution.a == -1.0f) {
       //     sm_payload.albedo = spEmittance;
            sm_payload.contribution.a = 0.f;
        }

        if (pv_envLightDescriptor.importance == 0)
            return;

        Vector3D direction = asVector3D(sm_ray.direction);
        float phi, theta;
        direction.toPolarYUp(&theta, &phi);

        float sinPhi, cosPhi;
        VLR::sincos(phi, &sinPhi, &cosPhi);
        Vector3D texCoord0Dir = normalize(Vector3D(-cosPhi, 0.0f, -sinPhi));
        ReferenceFrame shadingFrame;
        shadingFrame.x = texCoord0Dir;
        shadingFrame.z = -direction;
        shadingFrame.y = cross(shadingFrame.z, shadingFrame.x);

        SurfacePoint surfPt;
        surfPt.position = Point3D(direction.x, direction.y, direction.z);
        surfPt.shadingFrame = shadingFrame;
        surfPt.isPoint = false;
        surfPt.atInfinity = true;

        surfPt.geometricNormal = -direction;
        surfPt.u = phi;
        surfPt.v = theta;
        phi += pv_envLightDescriptor.body.asInfSphere.rotationPhi;
        phi = phi - std::floor(phi / (2 * M_PIf)) * 2 * M_PIf;
        surfPt.texCoord = TexCoord2D(phi / (2 * M_PIf), theta / M_PIf);

        float hypAreaPDF = evaluateEnvironmentAreaPDF(phi, theta);

        const SurfaceMaterialDescriptor matDesc = pv_materialDescriptorBuffer[pv_envLightDescriptor.materialIndex];
        EDF edf(matDesc, surfPt, sm_payload.wls);

        Vector3D dirOutLocal = surfPt.shadingFrame.toLocal(-asVector3D(sm_ray.direction));

        // implicit light sampling
        SampledSpectrum spEmittance = edf.evaluateEmittance();
        if (spEmittance.hasNonZero()) {
            SampledSpectrum Le = spEmittance * edf.evaluate(EDFQuery(), dirOutLocal);

            float MISWeight = 1.0f;
            if (!sm_payload.prevSampledType.isDelta() && sm_ray.ray_type != RayType::Primary) {
                float bsdfPDF = sm_payload.prevDirPDF;
                float dist2 = surfPt.calcSquaredDistance(asPoint3D(sm_ray.origin));
                float lightPDF = pv_envLightDescriptor.importance / getSumLightImportances() * hypAreaPDF * dist2 / std::fabs(dirOutLocal.z);
                MISWeight = (bsdfPDF * bsdfPDF) / (lightPDF * lightPDF + bsdfPDF * bsdfPDF);
            }

            sm_payload.contribution += sm_payload.alpha * Le * MISWeight;
			CLAMPINTENSITY(sm_payload.contribution, 100.f);
        }
    }


    // Common Ray Generation Program for All Camera Types
    RT_PROGRAM void pathTracing() {
        KernelRNG rng = pv_rngBuffer[sm_launchIndex];

        optix::float2 p = make_float2(sm_launchIndex.x + rng.getFloat0cTo1o(), sm_launchIndex.y + rng.getFloat0cTo1o());

        float selectWLPDF;
        WavelengthSamples wls = WavelengthSamples::createWithEqualOffsets(rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), &selectWLPDF);

        LensPosSample We0Sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        LensPosQueryResult We0Result;
        SampledSpectrum We0 = pv_progSampleLensPosition(wls, We0Sample, &We0Result);

        IDFSample We1Sample(p.x / pv_imageSize.x, p.y / pv_imageSize.y);
        IDFQueryResult We1Result;
        SampledSpectrum We1 = pv_progSampleIDF(We0Result.surfPt, wls, We1Sample, &We1Result);

        Vector3D rayDir = We0Result.surfPt.fromLocal(We1Result.dirLocal);
        SampledSpectrum alpha = (We0 * We1) * (We0Result.surfPt.calcCosTerm(rayDir) / (We0Result.areaPDF * We1Result.dirPDF * selectWLPDF));

        optix::Ray ray = optix::make_Ray(asOptiXType(We0Result.surfPt.position), asOptiXType(rayDir), RayType::Primary, 0.0f, FLT_MAX);

        Payload payload;
        payload.maxLengthTerminate = false;
        payload.rng = rng;
        payload.initImportance = alpha.importance(wls.selectedLambdaIndex());
        payload.wls = wls;
        payload.alpha = alpha;
        payload.contribution = SampledSpectrum::Zero();
		payload.contribution.a = -1.0f;
        //payload.normal = SampledSpectrum(0.0, 1.0, 0.0);
        //payload.albedo = SampledSpectrum(-1.f, -1.f, -1.f);

        const uint32_t MaxPathLength = 25;
        uint32_t pathLength = 0;
        while (true) {
            payload.terminate = true;
            ++pathLength;
            if (pathLength >= MaxPathLength)
                payload.maxLengthTerminate = true;
            rtTrace(pv_topGroup, ray, payload);

            if (payload.terminate)
                break;
            VLRAssert(pathLength < MaxPathLength, "Path should be terminated... Something went wrong...");

            ray = optix::make_Ray(asOptiXType(payload.origin), asOptiXType(payload.direction), RayType::Scattered, 0.0f, FLT_MAX);
        }
        pv_rngBuffer[sm_launchIndex] = payload.rng;
        if (!payload.contribution.allFinite()) {
           //	vlrprintf("Pass %u, (%u, %u): Not a finite value.\n", pv_numAccumFrames, sm_launchIndex.x, sm_launchIndex.y);
            return;
        }

        if (pv_numAccumFrames == 1) {
            pv_outputBuffer[sm_launchIndex].reset();
        //	pv_outputNormalBuffer[sm_launchIndex] = payload.normal;
        //	pv_outputAlbedoBuffer[sm_launchIndex] = payload.albedo;
        }
		//CLAMPINTENSITY(payload.contribution);
        pv_outputBuffer[sm_launchIndex].add(wls, payload.contribution);
    }

	

    // Exception Program
    RT_PROGRAM void exception() {
        //uint32_t code = rtGetExceptionCode();
        rtPrintExceptionDetails();
    }
}
